/*
  Uses N blocks with N threads
  SOR Stokes Flow with no slip b.c. on top/bottom and no flux b.c. on left/right written by Dmitriy Kats

  Inputs: N is the number of grid points in each direction, 
  	  	  mu is the viscosity
  	  	  Pdiff is the pressure drop in the x direction
  	  	  omega is the SOR factor
  	  	  toltau is the tolerance of the residual

  Outputs: The final velocities and pressure

 */



#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include<math.h>
#include <time.h>


//Kernels to udpate u, v, and p
//The inputs also considers if it is a red or black point udpate
__global__ void update_u(double* U, double* Uresid, double* P, double* Presid, double* FAC1, double* OMEGA, int RedorBlack);
__global__ void update_v(double* V, double* Vresid, double* P, double* Presid, double* FAC1, double* OMEGA, int RedorBlack);
__global__ void update_p(double* U, double* V, double* P, double* Presid, double* FAC1, double* OMEGA, double* Pdiff, int RedorBlack);

__device__ static int dev_N;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}



int main (int argc, char * argv[]){

	// Choose the GPU card
	hipDeviceProp_t prop;
	int dev;
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.multiProcessorCount = 13; 
	hipChooseDevice(&dev, &prop);
	hipSetDevice(dev);

	// Create the CUDA events that will be used for timing the kernel function
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Click, the timer has started running
	hipEventRecord(start, 0);



	int N;
	double mu, pdiff, omega, toltau;

	N=atoi(argv[1]);
	mu=atof(argv[2]);
	pdiff=atof(argv[3]);
	omega=atof(argv[4]);
	toltau=atof(argv[5]);

	double dx=1.0/((double)N-1.0);
	double fac1=dx/mu; //precompute the factor

	double residABSMAX = 99.0;
	int numberOfIterations=0;
	
	double* dev_fac1;
	double* dev_omega;
	double* dev_pdiff;

	double *dev_u, *dev_uresid;
	double *dev_v, *dev_vresid;
	double *dev_p, *dev_presid;
	
	//allocate memory for the velocities and pressure

	double *u = (double*)malloc(N*(N-1)*sizeof(double));
	double *uresid = (double*)malloc(N*(N-1)*sizeof(double));

	double *v = (double*)malloc((N-1)*N*sizeof(double));
	double *vresid = (double*)malloc((N-1)*N*sizeof(double));

	double *p = (double*)malloc((N+1)*(N-1)*sizeof(double));
	double *presid = (double*)malloc((N+1)*(N-1)*sizeof(double));
	
	//allocate Cuda memory
	hipMalloc((void**)&dev_fac1, sizeof(double));
	hipMalloc((void**)&dev_omega, sizeof(double));
	hipMalloc((void**)&dev_pdiff, sizeof(double));
	hipMalloc((void**)&dev_u, N*(N-1)*sizeof(double));
	hipMalloc((void**)&dev_uresid, N*(N-1)*sizeof(double));
	hipMalloc((void**)&dev_v, (N-1)*N*sizeof(double));
	hipMalloc((void**)&dev_vresid, (N-1)*N*sizeof(double));
	hipMalloc((void**)&dev_p, (N+1)*(N-1)*sizeof(double));
	hipMalloc((void**)&dev_presid, (N+1)*(N-1)*sizeof(double));

	//Intialize to zero
	int i, j;
	for(i=0; i<N; i++)
	{
		for(j=0; j<N-1; j++)
		{
			u[i+j*N]=0.0;  
			uresid[i+j*N]=0.0;
		}
	}

	for(i=0; i<N-1; i++)
	{
		for(j=0; j<N; j++)
		{
			v[i+j*(N-1)]=0.0;   
			vresid[i+j*(N-1)]=0.0;
		}
	}

	for(i=0; i<N+1; i++)
	{
		for(j=0; j<N-1; j++)
		{
			p[i+j*(N+1)]=0.0; 
			presid[i+j*(N+1)]=0.0;
		}
	}
	
	//Copy the values to the device
	hipMemcpy(dev_u, u, N*(N-1)*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_uresid, uresid, N*(N-1)*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_v, v, (N-1)*N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_vresid, vresid, (N-1)*N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_p, p, (N+1)*(N-1)*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_presid, presid, (N+1)*(N-1)*sizeof(double), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));
	hipMemcpy(dev_fac1, &fac1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_omega, &omega, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_pdiff, &pdiff, sizeof(double), hipMemcpyHostToDevice);

	
	dim3 meshDim(N,N); //This one will be for the velocities
	dim3 meshDim2(N+1,N); //This one will be for the pressure


	while(residABSMAX>=toltau)
	{
		residABSMAX=0.1*toltau;
		//Solve in the next six lines
		update_u<<<meshDim,1>>>(dev_u, dev_uresid, dev_p, dev_presid, dev_fac1, dev_omega, 0);
		update_u<<<meshDim,1>>>(dev_u, dev_uresid, dev_p, dev_presid,  dev_fac1, dev_omega, 1);
		update_v<<<meshDim,1>>>(dev_v, dev_vresid, dev_p, dev_presid,  dev_fac1, dev_omega, 0);
		update_v<<<meshDim,1>>>(dev_v, dev_vresid, dev_p, dev_presid, dev_fac1, dev_omega, 1);
		update_p<<<meshDim2,1>>>(dev_u, dev_v, dev_p, dev_presid,  dev_fac1, dev_omega, dev_pdiff, 0);
		update_p<<<meshDim2,1>>>(dev_u, dev_v, dev_p, dev_presid,  dev_fac1, dev_omega, dev_pdiff, 1);
		
		
		//This is slow but I ran out of time
		//Copy the residuals to the host to find the max residual
		hipMemcpy(uresid, dev_uresid, N*(N-1)*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(vresid, dev_vresid, (N-1)*N*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(presid, dev_presid, (N+1)*(N-1)*sizeof(double), hipMemcpyDeviceToHost);

		for(i=0; i<N; i++)
		{
			for(j=0; j<N-1; j++)
			{
				if(fabs(uresid[i+j*N])>residABSMAX)
				{
					residABSMAX=fabs(uresid[i+j*N]);
				}
			}
		}

		for(i=0; i<N-1; i++)
		{
			for(j=0; j<N; j++)
			{
				if(fabs(vresid[i+j*(N-1)])>residABSMAX)
				{
					residABSMAX=fabs(vresid[i+j*(N-1)]);
				}
			}
		}

		for(i=0; i<N+1; i++)
		{
			for(j=0; j<N-1; j++)
			{
				if(fabs(presid[i+j*(N+1)])>residABSMAX)
				{
					residABSMAX=fabs(presid[i+j*(N+1)]);
				}
			}
		}

		
		//Check for errors
		gpuErrchk(hipPeekAtLastError() );
		gpuErrchk(hipDeviceSynchronize() );

		numberOfIterations+=1;


		if (numberOfIterations>10000)
		{	//fail safe to save data and exit
			hipMemcpy(u, dev_u, N*(N-1)*sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(v, dev_v, (N-1)*N*sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(p, dev_p, (N+1)*(N-1)*sizeof(double), hipMemcpyDeviceToHost);

			printf("Reached fail safe. The max residual is %10e. The number of iterations is %i\n", residABSMAX, numberOfIterations);
			FILE *fpu = fopen("StokesU.out", "wb");
			fwrite(u, sizeof(double), N*(N-1), fpu);
			fclose (fpu);
			FILE *fpv = fopen("StokesV.out", "wb");
			fwrite(v, sizeof(double), (N-1)*N, fpv);
			fclose (fpv);
			FILE *fpP = fopen("StokesP.out", "wb");
			fwrite(p, sizeof(double), (N+1)*(N-1), fpP);
			fclose (fpP);

			hipFree(dev_u);
			hipFree(dev_uresid);
			hipFree(dev_v);
			hipFree(dev_vresid);
			hipFree(dev_p);
			hipFree(dev_presid);
			hipFree(dev_fac1);
			hipFree(dev_omega);
			hipFree(dev_pdiff);

			free(u);
			free(uresid);
			free(v);
			free(vresid);
			free(p);
			free(presid);

			return 0;
		}
	}

	hipMemcpy(u, dev_u, N*(N-1)*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(v, dev_v, (N-1)*N*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(p, dev_p, (N+1)*(N-1)*sizeof(double), hipMemcpyDeviceToHost);
	//export the data
	FILE *fpu = fopen("StokesU.out", "wb");
	fwrite(u, sizeof(double), N*(N-1), fpu);
	fclose (fpu);
	FILE *fpv = fopen("StokesV.out", "wb");
	fwrite(v, sizeof(double), (N-1)*N, fpv);
	fclose (fpv);
	FILE *fpP = fopen("StokesP.out", "wb");
	fwrite(p, sizeof(double), (N+1)*(N-1), fpP);
	fclose (fpP);

	//stop the timer
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// The elapsed time is computed by taking the difference between start and stop
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("N:%i omega:%f\n", N, omega);
	printf("The max residual is %10e and the number of iterations is %i\n", residABSMAX, numberOfIterations);
	printf("Time: %gms\n", elapsedTime);

	//clean up timer
	hipEventDestroy(start); 
	hipEventDestroy(stop);


	hipFree(dev_u);
	hipFree(dev_uresid);
	hipFree(dev_v);
	hipFree(dev_vresid);
	hipFree(dev_p);
	hipFree(dev_presid);
	hipFree(dev_fac1);
	hipFree(dev_omega);

	free(u);
	free(uresid);
	free(v);
	free(vresid);
	free(p);
	free(presid);

	return 0;
}

__global__ void update_u(double* U, double* Uresid, double* P, double* Presid, double* FAC1, double* OMEGA, int RorB)
{
	int EvenOrOdd=(blockIdx.x+blockIdx.y)%2;

	int u_ij00 = blockIdx.x + blockIdx.y * gridDim.x;
	int u_ijp0 = (blockIdx.x + 1)%gridDim.x + blockIdx.y * gridDim.x; //down for u
	int u_ijm0 = (blockIdx.x + gridDim.x - 1)%gridDim.x + blockIdx.y * gridDim.x; //up for u
	int u_ij0p = blockIdx.x + ((blockIdx.y + 1)%gridDim.y) * gridDim.x;             //east for u
	int u_ij0m = blockIdx.x + ((blockIdx.y + gridDim.y - 1)%gridDim.y) * gridDim.x; //west for u

	int p_ij00 = blockIdx.x + blockIdx.y * (gridDim.x+1);
	int p_ijp0 = (blockIdx.x + 1)%(gridDim.x+1) + blockIdx.y * (gridDim.x+1); //down for p
	//int p_ijm0 = (blockIdx.x + gridDim.x)%(gridDim.x+1) + blockIdx.y * (gridDim.x+1); //up for p
	//int p_ij0p = blockIdx.x + ((blockIdx.y + 1)%gridDim.y) *(gridDim.x+1);             //east for p
	//int p_ij0m = blockIdx.x + ((blockIdx.y + gridDim.y - 1)%gridDim.y) * (gridDim.x+1); //west for p

	//UPDATE INLET
	if (blockIdx.y==0 && blockIdx.x==0 && EvenOrOdd==RorB)
	{ 	 //Corner point
		Uresid[u_ij00]= (-U[u_ij00]+ U[u_ijp0])+(-3.0*U[u_ij00]+U[u_ij0p])-*FAC1*(P[p_ijp0]-P[p_ij00]);
		U[u_ij00]=U[u_ij00]+*OMEGA*Uresid[u_ij00]; 
	}

	if (blockIdx.y>0 && blockIdx.y<(dev_N-2) && blockIdx.x==0 && EvenOrOdd==RorB)
	{	  //Middle points
		Uresid[u_ij00]=(-U[u_ij00]+ U[u_ijp0])+(U[u_ij0m]-2.0*U[u_ij00]+U[u_ij0p])-*FAC1*(P[p_ijp0]-P[p_ij00]);
		U[u_ij00]=U[u_ij00]+*OMEGA*Uresid[u_ij00];  
	}

	if (blockIdx.y==(dev_N-2) && blockIdx.x==0 && EvenOrOdd==RorB)
	{	//Corner point 
		Uresid[u_ij00]= (-U[u_ij00]+ U[u_ijp0])+(U[u_ij0m]-3.0*U[u_ij00])-*FAC1*(P[p_ijp0]-P[p_ij00]);
		U[u_ij00]=U[u_ij00]+*OMEGA*Uresid[u_ij00]; 
	}

	//UPDATE BULK
	if (blockIdx.y==0 && blockIdx.x>0 && blockIdx.x<(dev_N-1)&& EvenOrOdd==RorB)
	{ // boundary condition
		Uresid[u_ij00]= (U[u_ijm0]-2.0*U[u_ij00]+ U[u_ijp0])+(-3.0*U[u_ij00]+U[u_ij0p])-*FAC1*(P[p_ijp0]-P[p_ij00]);
		U[u_ij00]=U[u_ij00]+*OMEGA*Uresid[u_ij00]; 
	}
	if (blockIdx.y>0 && blockIdx.y<(dev_N-2) && blockIdx.x>0 && blockIdx.x<(dev_N-1)&& EvenOrOdd==RorB)
	{ //interior
		Uresid[u_ij00]= (U[u_ijm0]-2.0*U[u_ij00]+ U[u_ijp0])+(U[u_ij0m]-2.0*U[u_ij00]+U[u_ij0p])-*FAC1*(P[p_ijp0]-P[p_ij00]);
		U[u_ij00]=U[u_ij00]+*OMEGA*Uresid[u_ij00]; 
	}
	if (blockIdx.y==(dev_N-2) && blockIdx.x>0 && blockIdx.x<(dev_N-1)&& EvenOrOdd==RorB)
	{ //boundary condition
		Uresid[u_ij00]= (U[u_ijm0]-2.0*U[u_ij00]+ U[u_ijp0])+(U[u_ij0m]-3.0*U[u_ij00])-*FAC1*(P[p_ijp0]-P[p_ij00]);
		U[u_ij00]=U[u_ij00]+*OMEGA*Uresid[u_ij00]; 
	}

	//Update Outlet
	if (blockIdx.y==0 && blockIdx.x==(dev_N-1)&& EvenOrOdd==RorB)
	{ //boundary condition
		Uresid[u_ij00]= (U[u_ijm0]-U[u_ij00])+(-3.0*U[u_ij00]+U[u_ij0p])-*FAC1*(P[p_ijp0]-P[p_ij00]);
		U[u_ij00]=U[u_ij00]+*OMEGA*Uresid[u_ij00]; 
	}
	if (blockIdx.y>0 && blockIdx.y<(dev_N-2) && blockIdx.x==(dev_N-1)&& EvenOrOdd==RorB)
	{ //middle points on outlet
		Uresid[u_ij00]= (U[u_ijm0]-U[u_ij00])+(U[u_ij0m]-2.0*U[u_ij00]+U[u_ij0p])-*FAC1*(P[p_ijp0]-P[p_ij00]);
		U[u_ij00]=U[u_ij00]+*OMEGA*Uresid[u_ij00]; 
	}
	if (blockIdx.y==(dev_N-2) && blockIdx.x==(dev_N-1)&& EvenOrOdd==RorB)
	{ //boundary node
		Uresid[u_ij00]= (U[u_ijm0]-U[u_ij00])+(U[u_ij0m]-3.0*U[u_ij00])-*FAC1*(P[p_ijp0]-P[p_ij00]);
		U[u_ij00]=U[u_ij00]+*OMEGA*Uresid[u_ij00]; 
	}

	__syncthreads();

}


__global__ void update_v(double* V, double* Vresid, double* P, double* Presid, double* FAC1, double* OMEGA, int RorB)
{
	int EvenOrOdd=(blockIdx.x+blockIdx.y)%2;

	int v_ij00 = blockIdx.x + blockIdx.y * (gridDim.x-1);
	int v_ijp0 = (blockIdx.x + 1)%(gridDim.x-1) + blockIdx.y * (gridDim.x-1); //down for v
	int v_ijm0 = (blockIdx.x + gridDim.x - 2)%(gridDim.x-1) + blockIdx.y * (gridDim.x-1); //up for v
	int v_ij0p = blockIdx.x + ((blockIdx.y + 1)%gridDim.y) * (gridDim.x-1);             //east for v
	int v_ij0m = blockIdx.x + ((blockIdx.y + gridDim.y - 1)%gridDim.y) * (gridDim.x-1); //west for v

	//int p_ij00 = blockIdx.x + blockIdx.y * (gridDim.x+1);
	int p_ijp0 = (blockIdx.x + 1)%(gridDim.x+1) + blockIdx.y * (gridDim.x+1); //down for p
	//int p_ijm0 = (blockIdx.x + gridDim.x)%(gridDim.x+1) + blockIdx.y * (gridDim.x+1); //up for p
	//int p_ij0p = blockIdx.x + ((blockIdx.y + 1)%gridDim.y) *(gridDim.x+1);             //east for p
	//int p_ij0m = blockIdx.x + ((blockIdx.y + gridDim.y - 1)%gridDim.y) * (gridDim.x+1); //west for p
	int p_ijpm = (blockIdx.x + 1)%(gridDim.x+1) + ((blockIdx.y + gridDim.y - 1)%gridDim.y) * (gridDim.x+1); //sw for p
	
	//Update inlet similarly to above
	if (blockIdx.y==0 && blockIdx.x==0 && EvenOrOdd==RorB)
	{   //no velocity boundary condition
		Vresid[v_ij00]= 0.0;
		V[v_ij00]=0.0;	
	}

	if (blockIdx.y>0 && blockIdx.y<(dev_N-1) && blockIdx.x==0 && EvenOrOdd==RorB)
	{	  
		Vresid[v_ij00]=(-V[v_ij00]+ V[v_ijp0])+(V[v_ij0m]-2.0*V[v_ij00]+V[v_ij0p])-*FAC1*(P[p_ijp0]-P[p_ijpm]);
		V[v_ij00]=V[v_ij00]+*OMEGA*Vresid[v_ij00];  
	}

	if (blockIdx.y==(dev_N-1) && blockIdx.x==0 && EvenOrOdd==0)
	{	  
		Vresid[v_ij00]= 0.0;
		V[v_ij00]=0.0;	 
	}

	//Update Bulk similarly to above
	if (blockIdx.y==0 && blockIdx.x>0 && blockIdx.x<(dev_N-2)&& EvenOrOdd==RorB)
	{
		Vresid[v_ij00]= 0.0;
		V[v_ij00]=0.0;
	}
	if (blockIdx.y>0 && blockIdx.y<(dev_N-1) && blockIdx.x>0 && blockIdx.x<(dev_N-2)&& EvenOrOdd==RorB)
	{
		Vresid[v_ij00]=(V[v_ijm0]-2.0*V[v_ij00]+ V[v_ijp0])+(V[v_ij0m]-2.0*V[v_ij00]+V[v_ij0p])-*FAC1*(P[p_ijp0]-P[p_ijpm]);
		V[v_ij00]=V[v_ij00]+*OMEGA*Vresid[v_ij00];
	}

	if (blockIdx.y==(dev_N-1) && blockIdx.x>0 && blockIdx.x<(dev_N-2)&& EvenOrOdd==RorB)
	{
		Vresid[v_ij00]=0.0;
		V[v_ij00]=0.0;
	}

	//Update Outlet
	if (blockIdx.y==0 && blockIdx.x==(dev_N-2)&& EvenOrOdd==RorB)
	{
		Vresid[v_ij00]= 0.0;
		V[v_ij00]=0.0;
	}
	if (blockIdx.y>0 && blockIdx.y<(dev_N-1) &&  blockIdx.x==(dev_N-2)&& EvenOrOdd==RorB)
	{
		Vresid[v_ij00]=(V[v_ijm0]-V[v_ij00])+(V[v_ij0m]-2.0*V[v_ij00]+V[v_ij0p])-*FAC1*(P[p_ijp0]-P[p_ijpm]);
		V[v_ij00]=V[v_ij00]+*OMEGA*Vresid[v_ij00];
	}
	if (blockIdx.y==(dev_N-1) && blockIdx.x==(dev_N-2)&& EvenOrOdd==RorB)
	{
		Vresid[v_ij00]= 0.0;
		V[v_ij00]=0.0;
	}
	__syncthreads();

}

__global__ void update_p(double* U, double* V, double* P, double* Presid, double* FAC1, double* OMEGA, double* Pdiff, int RorB)
{
	int EvenOrOdd=((int) (blockIdx.x+blockIdx.y)%2);

	int u_ij00 = blockIdx.x + blockIdx.y * (gridDim.x-1);
	int u_ijm0 = (blockIdx.x + gridDim.x - 2)%(gridDim.x-1) + blockIdx.y * (gridDim.x-1); //up for u

	int v_ijm0 = (blockIdx.x + gridDim.x - 3)%(gridDim.x-2) + blockIdx.y * (gridDim.x-2); //up for v
	int v_ijmp = (blockIdx.x + gridDim.x - 3)%(gridDim.x-2) + ((blockIdx.y + 1)%gridDim.y) * (gridDim.x-2);

	int p_ij00 = blockIdx.x + blockIdx.y * (gridDim.x);
	int p_ijp0 = (blockIdx.x + 1)%(gridDim.x) + blockIdx.y * (gridDim.x); //down for p
	int p_ijm0 = (blockIdx.x + gridDim.x-1)%(gridDim.x) + blockIdx.y * (gridDim.x); //up for p

	//Update the boundary with the right pressure drop
	if (blockIdx.y<(dev_N-1) && blockIdx.x==0 && EvenOrOdd==RorB)
	{	  
		Presid[p_ij00]=2.0*(*Pdiff)-P[p_ijp0]-P[p_ij00];
		P[p_ij00]=2.0*(*Pdiff)-P[p_ijp0];
	}
	//Update interior nodes
	if (blockIdx.y<(dev_N-1) && blockIdx.x>0 && blockIdx.x<(dev_N) && EvenOrOdd==RorB)
	{	  
		Presid[p_ij00]=-(U[u_ij00]-U[u_ijm0])-(V[v_ijmp]-V[v_ijm0]);
		P[p_ij00]=P[p_ij00]+*OMEGA*Presid[p_ij00];
	}
	//Update boundary conditions
	if (blockIdx.y<(dev_N-1) && blockIdx.x==(dev_N) && EvenOrOdd==RorB)
	{	  
		P[p_ij00]=-P[p_ijm0];
	}


	__syncthreads();
}
